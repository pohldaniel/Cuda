#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include "ray.h"
#include "vec3.h"
#include "hitable.h"
#include "hitable_list.h"
#include "sphere.h"
#include "camera.h"
#include "material.h"

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
	for (int i = 0; i < 50; i++) {
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			ray scattered;
			vec3 attenuation;
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else {
				return vec3(0.0, 0.0, 0.0);
			}
		}
		else {
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f*(unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void _rand_init(hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprand_init(1984, 0, 0, rand_state);
	}
}

__global__ void _render_init(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j*max_x + i;
	// Original: Each thread gets same seed, a different sequence number, no offset
	// hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
	// BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
	// performance improvement of about 2x!
	hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void _render(float *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j*max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);
	for (int s = 0; s < ns; s++) {
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		ray r = (*cam)->get_ray(u, v, &local_rand_state);
		col += color(r, world, &local_rand_state);
	}
	rand_state[pixel_index] = local_rand_state;
	col /= float(ns);
	col[0] = sqrt(col[0]);
	col[1] = sqrt(col[1]);
	col[2] = sqrt(col[2]);

	fb[pixel_index * 4 + 0] = col[0];
	fb[pixel_index * 4 + 1] = col[1];
	fb[pixel_index * 4 + 2] = col[2];
	fb[pixel_index * 4 + 3] = 1.0;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void _create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprandState local_rand_state = *rand_state;
		d_list[0] = new sphere(vec3(0, -1000.0, -1), 1000,
			new lambertian(vec3(0.5, 0.5, 0.5)));
		int i = 1;
		for (int a = -11; a < 11; a++) {
			for (int b = -11; b < 11; b++) {
				float choose_mat = RND;
				vec3 center(a + RND, 0.2, b + RND);
				if (choose_mat < 0.8f) {
					d_list[i++] = new sphere(center, 0.2,
						new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
				}
				else if (choose_mat < 0.95f) {
					d_list[i++] = new sphere(center, 0.2,
						new metal(vec3(0.5f*(1.0f + RND), 0.5f*(1.0f + RND), 0.5f*(1.0f + RND)), 0.5f*RND));
				}
				else {
					d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
				}
			}
		}
		d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
		d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
		d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
		*rand_state = local_rand_state;
		*d_world = new hitable_list(d_list, 22 * 22 + 1 + 3);

		vec3 lookfrom(13, 2, 3);
		vec3 lookat(0, 0, 0);
		float dist_to_focus = 10.0; (lookfrom - lookat).length();
		float aperture = 0.1;
		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			30.0,
			float(nx) / float(ny),
			aperture,
			dist_to_focus);
	}
}

__global__ void _free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
	for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
		delete ((sphere *)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}

extern "C" void render(dim3 blocks, dim3 threads, float *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
	_render << <blocks, threads >> >(fb, max_x, max_y, ns, cam, world, rand_state);
}

extern "C" void create_world(hitable **list, hitable **world, camera **camera, int nx, int ny, hiprandState *rand_state) {
	_create_world << <1, 1 >> >(list, world, camera, nx, ny, rand_state);
}

extern "C" void render_init(dim3 blocks, dim3 threads, int max_x, int max_y, hiprandState *rand_state) {	
	_render_init << <blocks, threads >> >(max_x, max_y, rand_state);
}

extern "C" void rand_init(hiprandState *rand_state) {
	_rand_init << <1, 1 >> >(rand_state);
}

extern "C" void free_world(hitable **list, hitable **world, camera **camera) {
	_free_world << <1, 1 >> >(list, world, camera);
}