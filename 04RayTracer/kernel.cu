#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ray.h"
#include "vec3.h"
#include "hitable.h"
#include "hitable_list.h"
#include "sphere.h"

__device__ vec3 color(const ray& r, hitable **world) {
	hit_record rec;
	if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
		return 0.5f*vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
	}
	else {
		vec3 unit_direction = unit_vector(r.direction());
		float t = 0.5f*(unit_direction.y() + 1.0f);
		return (1.0f - t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
	}
}

__global__ void _create_world(hitable **d_list, hitable **d_world) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		*(d_list) = new sphere(vec3(0, 0, -1), 0.5);
		*(d_list + 1) = new sphere(vec3(0, -100.5, -1), 100);
		*d_world = new hitable_list(d_list, 2);
	}
}

__global__ void free_world(hitable **d_list, hitable **d_world) {
	delete *(d_list);
	delete *(d_list + 1);
	delete *d_world;
}

__global__ void render(float *fb, int max_x, int max_y, vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin, hitable **world) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x * 4 + i * 4;
	float u = float(i) / float(max_x);
	float v = float(j) / float(max_y);
	ray r(origin, lower_left_corner + u*horizontal + v*vertical);
	vec3 c = color(r, world);

	fb[pixel_index + 0] = c.r();
	fb[pixel_index + 1] = c.g();
	fb[pixel_index + 2] = c.b();
	fb[pixel_index + 3] = 1.0;
}

extern "C" void create_image(dim3 blocks, dim3 threads, float *fb, int max_x, int max_y, vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin, hitable **world) {
	render << <blocks, threads >> >(fb, max_x, max_y, lower_left_corner, horizontal, vertical, origin, world);
}


extern "C" void create_world(hitable **list, hitable **world) {
	_create_world << <1, 1 >> >(list, world);
}